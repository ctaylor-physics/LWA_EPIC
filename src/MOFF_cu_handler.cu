#include "ex/MOFF_cu_handler.h"
#include "ex/constants.h"
#include "ex/cu_helpers.cuh"
#include "ex/fft_dx.cuh"
#include "ex/gridder.cuh"
#include "ex/types.hpp"
#include <hip/hip_cooperative_groups.h>
#include <hip/hip_fp16.h>
#include <iostream>
#include <nvml.h>

namespace cg = cooperative_groups;

void
MOFFCuHandler::reset_antpos(int p_nchan, float* p_antpos_ptr)
{
    cuda_check_err(hipSetDevice(m_device_id));
    if (is_antpos_set) {
        hipFree(m_antpos_cu);
    }
    auto nbytes = LWA_SV_NSTANDS * p_nchan * 3 * sizeof(float);
    hipMalloc(&m_antpos_cu, nbytes);
    hipMemcpy(m_antpos_cu, p_antpos_ptr, nbytes, hipMemcpyHostToDevice);
    is_antpos_set = true;
}

void
MOFFCuHandler::reset_phases(int p_nchan, float* p_phases_ptr)
{
    cuda_check_err(hipSetDevice(m_device_id));
    if (is_phases_set) {
        hipFree(m_phases_cu);
    }

    auto nbytes = LWA_SV_NSTANDS * p_nchan * LWA_SV_NPOLS * 2 * sizeof(float);
    hipMalloc(&m_phases_cu, nbytes);
    hipMemcpy(m_phases_cu, p_phases_ptr, nbytes, hipMemcpyHostToDevice);

    is_phases_set = true;
}

void
MOFFCuHandler::reset_gcf_tex(int p_gcf_tex_dim, float* p_gcf_2D_ptr)
{
    cuda_check_err(hipSetDevice(m_device_id));
    if (is_gcf_tex_set) {
        hipFreeArray(m_gcf_tex_arr);
        hipDestroyTextureObject(m_gcf_tex);
    }

    hipMallocArray(&m_gcf_tex_arr, &m_gcf_chan_desc, p_gcf_tex_dim, p_gcf_tex_dim);

    memset(&m_gcf_res_desc, 0, sizeof(m_gcf_res_desc));
    m_gcf_res_desc.resType = hipResourceTypeArray;
    m_gcf_res_desc.res.array.array = m_gcf_tex_arr;
    // Specify texture object parameters
    memset(&m_gcf_tex_desc, 0, sizeof(m_gcf_tex_desc));
    m_gcf_tex_desc.addressMode[0] = hipAddressModeClamp;
    m_gcf_tex_desc.addressMode[1] = hipAddressModeClamp;
    // m_gcf_tex_desc.filterMode = hipFilterModePoint;
    m_gcf_tex_desc.filterMode = hipFilterModeLinear;
    m_gcf_tex_desc.readMode = hipReadModeElementType;
    m_gcf_tex_desc.normalizedCoords = 0;

    std::cout << "copying gcf\n";
    const size_t spitch = p_gcf_tex_dim * sizeof(float);
    hipMemcpy2DToArray(m_gcf_tex_arr, 0, 0, p_gcf_2D_ptr, spitch, p_gcf_tex_dim * sizeof(float), p_gcf_tex_dim, hipMemcpyHostToDevice);

    std::cout << "texture set\n";
    hipCreateTextureObject(&m_gcf_tex, &m_gcf_res_desc, &m_gcf_tex_desc, NULL);

    is_gcf_tex_set = true;
    cuda_check_err(hipPeekAtLastError());
}

void
MOFFCuHandler::create_gulp_custreams()
{
    cuda_check_err(hipSetDevice(m_device_id));
    m_gulp_custreams.reset();
    m_gulp_custreams = std::make_unique<hipStream_t[]>(m_nstreams);
    for (int i = 0; i < m_nstreams; ++i) {
        hipStreamCreate(m_gulp_custreams.get() + i);
    }
}

void MOFFCuHandler::reset_gcf_elem(int p_nchan, int p_support, int p_chan0, float p_delta, int p_grid_size){
    cuda_check_err(hipSetDevice(m_device_id));
    if(is_m_gcf_elem_set){
        cuda_check_err(hipFree(m_gcf_elem));
        is_m_gcf_elem_set = false;
    }
    auto nelements_gcf = (p_support) * (p_support);
    auto nbytes = LWA_SV_NSTANDS * p_nchan * nelements_gcf * sizeof(float);
    cuda_check_err(hipMalloc(&m_gcf_elem, nbytes));
    is_m_gcf_elem_set=true;

    int block_size = int(MAX_THREADS_PER_BLOCK/nelements_gcf) * nelements_gcf;

    std::cout<<"Pre-computing GCF elements\n"<<p_support<<std::endl;
    compute_gcf_elements<<<p_nchan, block_size>>>(m_gcf_elem, m_antpos_cu, p_chan0, p_delta, m_gcf_tex,p_grid_size, (p_support), LWA_SV_NSTANDS);

    hipDeviceSynchronize();
    cuda_check_err(hipPeekAtLastError());
}

void MOFFCuHandler::get_correction_kernel(float* p_out_kernel, int p_support){
    cuda_check_err(hipSetDevice(m_device_id));
    if(m_nchan_in==0){
        std::cout<<"Number of input channels is not set. Unable to compute the averaged kernel\n";
        exit(-1);
    }
    int nbytes = p_support * p_support * m_nchan_in * sizeof(float);
    if(is_correction_kernel_set){
        hipFree(m_correction_kernel_d);
        is_correction_kernel_set = false;
    }
    cuda_check_err(hipMalloc(&m_correction_kernel_d, nbytes));
    is_correction_kernel_set = true;

    

    compute_avg_gridding_kernel<<<m_nchan_in, LWA_SV_NSTANDS>>>(m_gcf_elem, m_correction_kernel_d ,m_nchan_in, p_support);

    hipMemcpy(p_out_kernel, m_correction_kernel_d, nbytes, hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
    cuda_check_err(hipPeekAtLastError());
}
//void MOFFCuHandler::set_correction_grid(float* corr_grid);


void MOFFCuHandler::set_correction_grid(float* p_in_correction_grid, int p_grid_size, int p_nchan){
    cuda_check_err(hipSetDevice(m_device_id));
    int nbytes = p_grid_size * p_grid_size * p_nchan * sizeof(float);
    if(is_correction_grid_set){
        hipFree(m_correction_grid_d);
        is_correction_grid_set=false;
    }
    cuda_check_err(hipMalloc(&m_correction_grid_d, nbytes));
    is_correction_grid_set = true;

    cuda_check_err(hipMemcpy(m_correction_grid_d, p_in_correction_grid, nbytes, hipMemcpyHostToDevice));
    std::cout<<"FINE3\n";
    hipDeviceSynchronize();
    cuda_check_err(hipPeekAtLastError());
}

void
MOFFCuHandler::reset_data(int p_nchan, size_t p_nseq_per_gulp, float* p_antpos_ptr, float* p_phases_ptr)
{
    cuda_check_err(hipSetDevice(m_device_id));
    m_nseq_per_gulp = p_nseq_per_gulp;
    m_nchan_in = p_nchan;

    std::cout << "GPU resetting antpos\n";
    reset_antpos(p_nchan, p_antpos_ptr);
    cuda_check_err(hipPeekAtLastError());

    std::cout << "GPU resetting phases\n";
    reset_phases(p_nchan, p_phases_ptr);
    cuda_check_err(hipPeekAtLastError());

    hipDeviceSynchronize();
    cuda_check_err(hipPeekAtLastError());
}

void
MOFFCuHandler::set_imaging_kernel()
{   int smemSize;
    hipDeviceGetAttribute(&smemSize, hipDeviceAttributeMaxSharedMemoryPerBlock, m_device_id);
    std::cout<<"Max shared memory per block: "<<smemSize<<" bytes\n";
    cuda_check_err(hipSetDevice(m_device_id));
    // assert(m_out_img_desc.img_size == HALF);
    if (m_out_img_desc.img_size == HALF) {
        std::cout<<"Setting the imaging kernel to 64x64\n";
        std::cout<<"Shared memory size: "<<FFT64x64::shared_memory_size<<" bytes\n";
        std::cout<<FFT64x64::block_dim.x<<" "<<FFT64x64::block_dim.y<<"\n";
        m_imaging_kernel = get_imaging_kernel<FFT64x64>(m_support_size);
        hipFuncSetAttribute(reinterpret_cast<const void*>(
          m_imaging_kernel),
          hipFuncAttributeMaxDynamicSharedMemorySize,
          FFT64x64::shared_memory_size*2);
        m_img_block_dim = FFT64x64::block_dim;
        m_shared_mem_size = FFT64x64::shared_memory_size*2;
    } else {
         std::cout<<"Setting the imaging kernel to 128x128\n";
        std::cout<<"Shared memory size: "<<FFT128x128::shared_memory_size<<" bytes "<<FFT128x128::elements_per_thread<<"\n";
        std::cout<<FFT64x64::block_dim.x<<" "<<FFT128x128::block_dim.y<<"\n";

        m_imaging_kernel = get_imaging_kernel<FFT128x128>(m_support_size);
        
        hipFuncSetAttribute(reinterpret_cast<const void*>(
          m_imaging_kernel),
          hipFuncAttributeMaxDynamicSharedMemorySize,
          FFT128x128::shared_memory_size*1.5);
        m_img_block_dim = FFT128x128::block_dim;
        m_shared_mem_size = FFT128x128::shared_memory_size*1.5;
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(m_imaging_kernel), hipFuncCachePreferL1);
    }
}

void
MOFFCuHandler::allocate_f_eng_gpu(size_t nbytes)
{
    cuda_check_err(hipSetDevice(m_device_id));
    if (is_f_eng_cu_allocated) {
        hipFree(m_f_eng_cu);
        is_f_eng_cu_allocated = false;
    }
    hipMalloc(&m_f_eng_cu, nbytes);
    m_f_eng_bytes = nbytes;
    is_f_eng_cu_allocated = true;
}

void
MOFFCuHandler::allocate_out_img(size_t p_nbytes)
{
    cuda_check_err(hipSetDevice(m_device_id));
    if (is_out_mem_set) {
        hipFree(m_output_cu);
        is_out_mem_set = false;
    }
    hipMalloc(&m_output_cu, p_nbytes);
    is_out_mem_set = true;
    m_out_img_bytes = p_nbytes;
}

void
MOFFCuHandler::set_img_grid_dim()
{
    cuda_check_err(hipSetDevice(m_device_id));
    assert((void("Number of channels per stream cannot be zero"), m_nchan_per_stream > 0));
    if (m_nchan_per_stream > 0) {
        m_img_grid_dim = dim3(m_nchan_per_stream, 1, 1);
    }
}

void
MOFFCuHandler::process_gulp(uint8_t* p_data_ptr, float* p_out_ptr, bool p_first, bool p_last, int p_chan0, float p_delta)
{
    cuda_check_err(hipSetDevice(m_device_id));
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    std::cout<<"FEng bytes per stream: "<<m_nbytes_f_eng_per_stream<<". OutImg bytes per stream: "<<m_nbytes_out_img_per_stream<<". Chan per stream: "<<m_nchan_per_stream<<". NStreams: "<<m_nstreams<<"\n";
    std::cout<<"Nseq per gulp: "<<m_nseq_per_gulp<<"\n";
    for (int i = 0; i < m_nstreams; ++i) {
        int f_eng_dat_offset = i * m_nbytes_f_eng_per_stream;
        int output_img_offset = i * m_nbytes_out_img_per_stream/sizeof(float);
        auto stream_i = *(m_gulp_custreams.get() + i);
        int chan_offset = i * m_nchan_per_stream;

        void* args[] = {
            &m_f_eng_cu, &m_antpos_cu, &m_phases_cu, &m_nseq_per_gulp, &m_nchan_in, &m_gcf_tex, &m_output_cu, &chan_offset, &p_first, &m_gcf_elem, &m_correction_grid_d
        };

        cuda_check_err(
          hipMemcpyAsync(
            (void*)(m_f_eng_cu + f_eng_dat_offset),
            (void*)(p_data_ptr + f_eng_dat_offset),
            m_nbytes_f_eng_per_stream,
            hipMemcpyHostToDevice,
            stream_i));
        std::cout<<"Launching the kernel\n";
        if(m_imaging_kernel==nullptr){
            std::cout<<"Null imaging kernel\n";
        }
        std::cout<<m_img_grid_dim.x<<" "<<m_img_grid_dim.y<<" "<<m_img_block_dim.x<<" "<<m_img_block_dim.y<<" "<<m_shared_mem_size<<std::endl;
        cuda_check_err(hipLaunchKernel(reinterpret_cast<const void*>(m_imaging_kernel), m_img_grid_dim, m_img_block_dim, args, m_shared_mem_size, stream_i));

        std::cout<<"chan0: "<<p_chan0<<" delta: "<<p_delta<<"\n";

        std::cout<<i<<" "<<output_img_offset<<" "<<"\n";
        if (p_last) {
            cuda_check_err(hipMemcpyAsync((void*)(p_out_ptr + output_img_offset), (void*)(m_output_cu + output_img_offset), m_nbytes_out_img_per_stream, hipMemcpyDeviceToHost, stream_i));
        }
    }

    if (p_last) {
        std::cout<<"Syncing the kernels\n";
        for (int i = 0; i < m_nstreams; ++i) {
            cuda_check_err(hipStreamSynchronize(*(m_gulp_custreams.get() + i)));
        }
        std::cout<<"Syncing done\n";
        cuda_check_err(hipPeekAtLastError());
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Gulp processing time (ms): " << milliseconds << std::endl;

}

void
MOFFCuHandler::destroy_textures(hipArray_t& p_tex_arr, hipTextureObject_t& p_tex_obj)
{
    cuda_check_err(hipSetDevice(m_device_id));
    hipFreeArray(p_tex_arr);
    hipDestroyTextureObject(p_tex_obj);
}

MOFFCuHandler::~MOFFCuHandler()
{
    cuda_check_err(hipSetDevice(m_device_id));
    // destroy_textures(m_antpos_tex_arr, m_antpos_tex);
    // destroy_textures(m_phases_tex_arr, m_phases_tex);
    if (is_antpos_set) {
        hipFree(m_antpos_cu);
    }
    if (is_phases_set) {
        hipFree(m_phases_cu);
    }

    if(is_correction_kernel_set){
        hipFree(m_correction_kernel_d);
    }

    if(is_correction_grid_set){
        hipFree(m_correction_grid_d);
    }

    if(is_m_gcf_elem_set){
        hipFree(m_gcf_elem);
    }

    if(is_out_mem_set){
        hipFree(m_output_cu);
    }

    if(is_f_eng_cu_allocated){
        hipFree(m_f_eng_cu);
    }
}
