#include "ex/MOFF_cu_handler.h"
#include "ex/constants.h"
#include "ex/cu_helpers.cuh"
#include "ex/fft_dx.cuh"
#include "ex/gridder.cuh"
#include "ex/types.hpp"
#include <hip/hip_cooperative_groups.h>
#include <hip/hip_fp16.h>
#include <iostream>
#include <nvml.h>

namespace cg = cooperative_groups;

void
MOFFCuHandler::reset_antpos(int p_nchan, float* p_antpos_ptr)
{
    hipSetDevice(m_device_id);
    if (is_antpos_set) {
        hipFree(m_antpos_cu);
    }
    auto nbytes = LWA_SV_NSTANDS * p_nchan * 3 * sizeof(float);
    hipMalloc(&m_antpos_cu, nbytes);
    hipMemcpy(m_antpos_cu, p_antpos_ptr, nbytes, hipMemcpyHostToDevice);
    is_antpos_set = true;
}

void
MOFFCuHandler::reset_phases(int p_nchan, float* p_phases_ptr)
{
    hipSetDevice(m_device_id);
    if (is_phases_set) {
        hipFree(m_phases_cu);
    }

    auto nbytes = LWA_SV_NSTANDS * p_nchan * LWA_SV_NPOLS * 2 * sizeof(float);
    hipMalloc(&m_phases_cu, nbytes);
    hipMemcpy(m_phases_cu, p_phases_ptr, nbytes, hipMemcpyHostToDevice);

    is_phases_set = true;
}

void
MOFFCuHandler::reset_gcf_tex(int p_gcf_tex_dim, float* p_gcf_2D_ptr)
{
    hipSetDevice(m_device_id);
    if (is_gcf_tex_set) {
        hipFreeArray(m_gcf_tex_arr);
        hipDestroyTextureObject(m_gcf_tex);
    }

    hipMallocArray(&m_gcf_tex_arr, &m_gcf_chan_desc, p_gcf_tex_dim, p_gcf_tex_dim);

    memset(&m_gcf_res_desc, 0, sizeof(m_gcf_res_desc));
    m_gcf_res_desc.resType = hipResourceTypeArray;
    m_gcf_res_desc.res.array.array = m_gcf_tex_arr;
    // Specify texture object parameters
    // struct hipTextureDesc tex_desc;
    memset(&m_gcf_tex_desc, 0, sizeof(m_gcf_tex_desc));
    m_gcf_tex_desc.addressMode[0] = hipAddressModeClamp;
    m_gcf_tex_desc.addressMode[1] = hipAddressModeClamp;
    // m_gcf_tex_desc.filterMode = hipFilterModePoint; //hipFilterModeLinear;
    m_gcf_tex_desc.filterMode = hipFilterModeLinear;
    m_gcf_tex_desc.readMode = hipReadModeElementType;
    m_gcf_tex_desc.normalizedCoords = 0;

    // m_gcf_res_desc.res.array.array = m_gcf_tex_arr;
    std::cout << "copying gcf\n";
    const size_t spitch = p_gcf_tex_dim * sizeof(float);
    hipMemcpy2DToArray(m_gcf_tex_arr, 0, 0, p_gcf_2D_ptr, spitch, p_gcf_tex_dim * sizeof(float), p_gcf_tex_dim, hipMemcpyHostToDevice);

    std::cout << "texture set\n";
    hipCreateTextureObject(&m_gcf_tex, &m_gcf_res_desc, &m_gcf_tex_desc, NULL);

    is_gcf_tex_set = true;
    cuda_check_err(hipPeekAtLastError());
}

void
MOFFCuHandler::create_gulp_custreams()
{
    hipSetDevice(m_device_id);
    m_gulp_custreams.reset();
    m_gulp_custreams = std::make_unique<hipStream_t[]>(m_nstreams);
    for (int i = 0; i < m_nstreams; ++i) {
        hipStreamCreate(m_gulp_custreams.get() + i);
    }
}

void
MOFFCuHandler::reset_data(int p_nchan, size_t p_nseq_per_gulp, float* p_antpos_ptr, float* p_phases_ptr)
{
    hipSetDevice(m_device_id);
    m_nseq_per_gulp = p_nseq_per_gulp;
    m_nchan_in = p_nchan;
    std::cout << "GPU resetting antpos\n";
    reset_antpos(p_nchan, p_antpos_ptr);
    std::cout << "GPU resetting phases\n";
    cuda_check_err(hipPeekAtLastError());

    reset_phases(p_nchan, p_phases_ptr);
    cuda_check_err(hipPeekAtLastError());

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    // test_gcf_texture<<<1, 1>>>(m_gcf_tex);
    cuda_check_err(hipPeekAtLastError());
    hipEventRecord(stop);
    cuda_check_err(hipPeekAtLastError());
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Single grid time: " << milliseconds << std::endl;
    hipDeviceSynchronize();
    cuda_check_err(hipPeekAtLastError());
}

void
MOFFCuHandler::set_imaging_kernel()
{   int smemSize;
    hipDeviceGetAttribute(&smemSize, hipDeviceAttributeMaxSharedMemoryPerBlock, m_device_id);
    std::cout<<"Max shared memory per block: "<<smemSize<<" bytes\n";
    hipSetDevice(m_device_id);
    // assert(m_out_img_desc.img_size == HALF);
    if (m_out_img_desc.img_size == HALF) {
        std::cout<<"Setting the imaging kernel to 64x64\n";
        std::cout<<"Shared memory size: "<<FFT64x64::shared_memory_size<<" bytes\n";
        std::cout<<FFT64x64::block_dim.x<<" "<<FFT64x64::block_dim.y<<"\n";
        m_imaging_kernel = (void*)(block_fft_kernel<FFT64x64>);
        hipFuncSetAttribute(reinterpret_cast<const void*>(
          m_imaging_kernel),
          hipFuncAttributeMaxDynamicSharedMemorySize,
          FFT64x64::shared_memory_size*2);
        m_img_block_dim = FFT64x64::block_dim;
        m_shared_mem_size = FFT64x64::shared_memory_size*2;
    } else {
         std::cout<<"Setting the imaging kernel to 128x128\n";
        std::cout<<"Shared memory size: "<<FFT128x128::shared_memory_size<<" bytes "<<FFT128x128::elements_per_thread<<"\n";
        std::cout<<FFT64x64::block_dim.x<<" "<<FFT128x128::block_dim.y<<"\n";
        m_imaging_kernel = (void*)(block_fft_kernel<FFT128x128>);
        hipFuncSetAttribute(reinterpret_cast<const void*>(
          m_imaging_kernel),
          hipFuncAttributeMaxDynamicSharedMemorySize,
          FFT128x128::shared_memory_size*1.5);
        m_img_block_dim = FFT128x128::block_dim;
        m_shared_mem_size = FFT128x128::shared_memory_size*1.5;
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(m_imaging_kernel), hipFuncCachePreferL1);
    }
}

void
MOFFCuHandler::allocate_f_eng_gpu(size_t nbytes)
{
    hipSetDevice(m_device_id);
    if (is_f_eng_cu_allocated) {
        hipFree(m_f_eng_cu);
        is_f_eng_cu_allocated = false;
    }
    hipMalloc(&m_f_eng_cu, nbytes);
    m_f_eng_bytes = nbytes;
    is_f_eng_cu_allocated = true;
}

void
MOFFCuHandler::allocate_out_img(size_t p_nbytes)
{
    hipSetDevice(m_device_id);
    if (is_out_mem_set) {
        hipFree(m_output_cu);
        is_out_mem_set = false;
    }
    hipMalloc(&m_output_cu, p_nbytes);
    is_out_mem_set = true;
    m_out_img_bytes = p_nbytes;
}

void
MOFFCuHandler::set_img_grid_dim()
{
    hipSetDevice(m_device_id);
    assert((void("Number of channels per stream cannot be zero"), m_nchan_per_stream > 0));
    if (m_nchan_per_stream > 0) {
        m_img_grid_dim = dim3(m_nchan_per_stream, 1, 1);
    }
}

void
MOFFCuHandler::process_gulp(uint8_t* p_data_ptr, float* p_out_ptr, bool p_first, bool p_last, int p_chan0, float p_delta)
{
    hipSetDevice(m_device_id);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    std::cout<<"FEng bytes per stream: "<<m_nbytes_f_eng_per_stream<<". OutImg bytes per stream: "<<m_nbytes_out_img_per_stream<<". Chan per stream: "<<m_nchan_per_stream<<". NStreams: "<<m_nstreams<<"\n";
    std::cout<<"Nseq per gulp: "<<m_nseq_per_gulp<<"\n";
    for (int i = 0; i < m_nstreams; ++i) {
        int f_eng_dat_offset = i * m_nbytes_f_eng_per_stream;
        int output_img_offset = i * m_nbytes_out_img_per_stream/sizeof(float);
        auto stream_i = *(m_gulp_custreams.get() + i);
        int chan_offset = i * m_nchan_per_stream;

        void* args[] = {
            &m_f_eng_cu, &m_antpos_cu, &m_phases_cu, &m_nseq_per_gulp, &m_nchan_in, &m_gcf_tex, &m_output_cu, &chan_offset, &p_first, &p_chan0, &p_delta
        };

        cuda_check_err(
          hipMemcpyAsync(
            (void*)(m_f_eng_cu + f_eng_dat_offset),
            (void*)(p_data_ptr + f_eng_dat_offset),
            m_nbytes_f_eng_per_stream,
            hipMemcpyHostToDevice,
            stream_i));
        std::cout<<"Launching the kernel\n";
        if(m_imaging_kernel==nullptr){
            std::cout<<"Null imaging kernel\n";
        }
        std::cout<<m_img_grid_dim.x<<" "<<m_img_grid_dim.y<<" "<<m_img_block_dim.x<<" "<<m_img_block_dim.y<<" "<<m_shared_mem_size<<std::endl;
        cuda_check_err(hipLaunchKernel(reinterpret_cast<const void*>(m_imaging_kernel), m_img_grid_dim, m_img_block_dim, args, m_shared_mem_size, stream_i));

        std::cout<<"chan0: "<<p_chan0<<" delta: "<<p_delta<<"\n";

        std::cout<<i<<" "<<output_img_offset<<" "<<"\n";
        if (p_last) {
            cuda_check_err(hipMemcpyAsync((void*)(p_out_ptr + output_img_offset), (void*)(m_output_cu + output_img_offset), m_nbytes_out_img_per_stream, hipMemcpyDeviceToHost, stream_i));
        }
    }

    if (p_last) {
        std::cout<<"Syncing the kernels\n";
        for (int i = 0; i < m_nstreams; ++i) {
            cuda_check_err(hipStreamSynchronize(*(m_gulp_custreams.get() + i)));
        }
        std::cout<<"Syncing done\n";
        cuda_check_err(hipPeekAtLastError());
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Gulp processing time (ms): " << milliseconds << std::endl;

}

void
MOFFCuHandler::destroy_textures(hipArray_t& p_tex_arr, hipTextureObject_t& p_tex_obj)
{
    hipSetDevice(m_device_id);
    hipFreeArray(p_tex_arr);
    hipDestroyTextureObject(p_tex_obj);
}

MOFFCuHandler::~MOFFCuHandler()
{
    hipSetDevice(m_device_id);
    // destroy_textures(m_antpos_tex_arr, m_antpos_tex);
    // destroy_textures(m_phases_tex_arr, m_phases_tex);
    if (is_antpos_set) {
        hipFree(m_antpos_cu);
    }
    if (is_phases_set) {
        hipFree(m_phases_cu);
    }
}
